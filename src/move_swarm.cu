#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <boost/lexical_cast.hpp>
#include <math.h>

#include "ros/ros.h"
#include "gazebo_msgs/ModelState.h"

#include "hip/hip_runtime.h"

int main(int argc, char **argv)
{
	ros::init(argc, argv, "move_swarm");
	ros::NodeHandle node;

	ros::Publisher *swarm_pub;
	gazebo_msgs::ModelState *swarm_msg;

	swarm_pub = (ros::Publisher *)malloc(100*sizeof(ros::Publisher));
	swarm_msg = (gazebo_msgs::ModelState *)malloc(100*sizeof(gazebo_msgs::ModelState));

	for (int i = 0; i < 100; i++)
	{
		swarm_pub[i] = node.advertise<gazebo_msgs::ModelState>("/gazebo/set_model_state", 1);
		swarm_msg[i].model_name = "quadrotor" + boost::lexical_cast<std::string>(i);
	}

	int t = 0;
	while (ros::ok())
	{
		for (int i = 0; i < 100; i++)
		{
			swarm_msg[i].pose.position.x = 2.5*sin(t) + i/10;
			swarm_msg[i].pose.position.y = 2.5*sin(t) + i%10;
			swarm_msg[i].pose.position.z = 2;

			swarm_pub[i].publish(swarm_msg[i]);
		}
		t += 0.001;
	}

	free(swarm_pub); free(swarm_msg);
}

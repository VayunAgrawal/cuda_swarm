#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <boost/lexical_cast.hpp>
#include <math.h>

#include "ros/ros.h"
#include "gazebo_msgs/ModelState.h"
#include "tf/tf.h"

#include "hip/hip_runtime.h"

int main(int argc, char **argv)
{
	ros::init(argc, argv, "move_swarm");
	ros::NodeHandle node;

	ros::Publisher *swarm_pub = new ros::Publisher[100];
	gazebo_msgs::ModelState *swarm_msg = new gazebo_msgs::ModelState[100];

	for (int i = 0; i < 100; i++)
	{
		swarm_pub[i] = node.advertise<gazebo_msgs::ModelState>("/gazebo/set_model_state", 1);
		swarm_msg[i].model_name = "quadrotor" + boost::lexical_cast<std::string>(i);
	}

	float t = 0;
	tf::Quaternion quaternion;
	while (ros::ok())
	{
		for (int i = 0; i < 100; i++)
		{
			quaternion = tf::createQuaternionFromRPY(-0.5*cos(t), 0.5*cos(t), 0);
			swarm_msg[i].pose.position.x = sin(t) + i/10;
			swarm_msg[i].pose.position.y = sin(t) + i%10;
			swarm_msg[i].pose.position.z = sin(t+i) + 2;

			swarm_msg[i].pose.orientation.x = quaternion[0];
			swarm_msg[i].pose.orientation.y = quaternion[1];
			swarm_msg[i].pose.orientation.z = quaternion[2];
			swarm_msg[i].pose.orientation.w = quaternion[3];

			swarm_pub[i].publish(swarm_msg[i]);
		}
		t += 0.001;
	}

	delete[] swarm_pub; 
	delete[] swarm_msg;
}

#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <boost/lexical_cast.hpp>
#include <math.h>

#include "ros/ros.h"
#include "gazebo_msgs/ModelState.h"
#include "tf/tf.h"

#include "hip/hip_runtime.h"

__global__ void trajectories(gazebo_msgs::ModelState *msg, tf::Quaternion *quat, float t)
{

}

int main(int argc, char **argv)
{
	ros::init(argc, argv, "move_swarm");
	ros::NodeHandle node;

	ros::Publisher *swarm_pub = new ros::Publisher[100];
	gazebo_msgs::ModelState *swarm_msg = new gazebo_msgs::ModelState[100];

	for (int i = 0; i < 100; i++)
	{
		swarm_pub[i] = node.advertise<gazebo_msgs::ModelState>("/gazebo/set_model_state", 1);
		swarm_msg[i].model_name = "quadrotor" + boost::lexical_cast<std::string>(i);
	}

	hipError_t error;
	gazebo_msgs::ModelState *d_msg;
	if ((error = hipMalloc((void **)&d_msg, 100*sizeof(gazebo_msgs::ModelState))) != hipSuccess)
	{
		printf("Error allocating d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	if ((error = hipMemcpy(d_msg, swarm_msg, 100*sizeof(gazebo_msgs::ModelState), hipMemcpyHostToDevice)) != hipSuccess)
	{
		printf("Error copying a to d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	float t = 0;
	tf::Quaternion quaternion;
	while (ros::ok())
	{
		quaternion = tf::createQuaternionFromRPY(-0.5*cos(t), 0.5*cos(t), 0);
		
		for (int i = 0; i < 100; i++)
		{
			swarm_msg[i].pose.position.x = sin(t) + i/10;
			swarm_msg[i].pose.position.y = sin(t) + i%10;
			swarm_msg[i].pose.position.z = sin(t+i) + 2;

			swarm_msg[i].pose.orientation.x = quaternion[0];
			swarm_msg[i].pose.orientation.y = quaternion[1];
			swarm_msg[i].pose.orientation.z = quaternion[2];
			swarm_msg[i].pose.orientation.w = quaternion[3];

			swarm_pub[i].publish(swarm_msg[i]);
		}
		t += 0.001;
	}

	delete[] swarm_pub; 
	delete[] swarm_msg;
}

#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <boost/lexical_cast.hpp>
#include <math.h>
#include <time.h>

#include "ros/ros.h"
#include "gazebo_msgs/ModelState.h"
#include "tf/tf.h"

#include "hip/hip_runtime.h"

__global__ void trajectories(gazebo_msgs::ModelState *msg, float x, float y, float z, float w, float t)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	msg[i].pose.position.x = sin(t) + i/10;
	msg[i].pose.position.y = sin(t) + i%10;
	msg[i].pose.position.z = sin(t+i) + 2;

	msg[i].pose.orientation.x = x;
	msg[i].pose.orientation.y = y;
	msg[i].pose.orientation.z = z;
	msg[i].pose.orientation.w = w;
}

int main(int argc, char **argv)
{
	ros::init(argc, argv, "move_swarm");
	ros::NodeHandle node;

	ros::Publisher swarm_pub = node.advertise<gazebo_msgs::ModelState>("/gazebo/set_model_state", 100);
	gazebo_msgs::ModelState *swarm_msg = new gazebo_msgs::ModelState[100];

	for (int i = 0; i < 100; i++)
	{
		swarm_msg[i].model_name = "quadrotor" + boost::lexical_cast<std::string>(i);
	}

	long msg_size = 100*sizeof(gazebo_msgs::ModelState);
	gazebo_msgs::ModelState *d_msg;

	hipError_t error;
	if ((error = hipMalloc((void **)&d_msg, msg_size)) != hipSuccess)
	{
		printf("Error allocating d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	if ((error = hipMemcpy(d_msg, swarm_msg, msg_size, hipMemcpyHostToDevice)) != hipSuccess)
	{
		printf("Error copying a to d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	clock_t initial, final;
	float t = 0;
	tf::Quaternion q;
	while (ros::ok())
	{
		q = tf::createQuaternionFromRPY(-0.5*cos(t), 0.5*cos(t), 0);

		initial = clock();
		trajectories<<<100,512>>>(d_msg, q[0], q[1], q[2], q[3], t);
		hipMemcpy(swarm_msg, d_msg, msg_size, hipMemcpyDeviceToHost);
		for (int i = 0; i < 100; i++) swarm_pub.publish(swarm_msg[i]);
		final = clock();

		std::cout << "Time taken by GPU: " << (double)(final-initial)/CLOCKS_PER_SEC << std::endl;

		initial = clock();
		for (int i = 0; i < 100; i++)
		{
			swarm_msg[i].pose.position.x = sin(t) + i/10;
			swarm_msg[i].pose.position.y = sin(t) + i%10;
			swarm_msg[i].pose.position.z = sin(t+i) + 2;

			swarm_msg[i].pose.orientation.x = q[0];
			swarm_msg[i].pose.orientation.y = q[1];
			swarm_msg[i].pose.orientation.z = q[2];
			swarm_msg[i].pose.orientation.w = q[3];

			swarm_pub.publish(swarm_msg[i]);
		}
		final = clock();

		std::cout << "Time taken by CPU: " << (double)(final-initial)/CLOCKS_PER_SEC << std::endl;
		
		t += 0.001;
	}

	delete[] swarm_msg;
	hipFree(d_msg);
}

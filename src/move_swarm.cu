#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <boost/lexical_cast.hpp>
#include <math.h>

#include "ros/ros.h"
#include "gazebo_msgs/ModelState.h"
#include "tf/tf.h"

#include "hip/hip_runtime.h"

__global__ void trajectories(gazebo_msgs::ModelState *msg, float x, float y, float z, float w, float t)
{
	int i = threadIdx.x;

	msg[i].pose.position.x = sin(t) + i/10;
	msg[i].pose.position.y = sin(t) + i%10;
	msg[i].pose.position.z = sin(t+i) + 2;

	msg[i].pose.orientation.x = x;
	msg[i].pose.orientation.y = y;
	msg[i].pose.orientation.z = z;
	msg[i].pose.orientation.w = w;
}

int main(int argc, char **argv)
{
	ros::init(argc, argv, "move_swarm");
	ros::NodeHandle node;

	ros::Publisher swarm_pub = node.advertise<gazebo_msgs::ModelState>("/gazebo/set_model_state", 100);
	gazebo_msgs::ModelState *swarm_msg = new gazebo_msgs::ModelState[100];

	for (int i = 0; i < 100; i++)
	{
		swarm_msg[i].model_name = "quadrotor" + boost::lexical_cast<std::string>(i);
	}

	long size = 100*sizeof(gazebo_msgs::ModelState);
	hipError_t error;
	gazebo_msgs::ModelState *d_msg;
	if ((error = hipMalloc((void **)&d_msg, size)) != hipSuccess)
	{
		printf("Error allocating d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	if ((error = hipMemcpy(d_msg, swarm_msg, size, hipMemcpyHostToDevice)) != hipSuccess)
	{
		printf("Error copying a to d_a: %s in %s on line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	float t = 0;
	tf::Quaternion q;
	while (ros::ok())
	{
		q = tf::createQuaternionFromRPY(-0.5*cos(t), 0.5*cos(t), 0);
		trajectories<<<1,100>>>(d_msg, q[0], q[1], q[2], q[3], t);
		hipMemcpy(swarm_msg, d_msg, size, hipMemcpyDeviceToHost);
		for (int i = 0; i < 100; i++) swarm_pub.publish(swarm_msg[i]);
		t += 0.001;
	}

	delete[] swarm_msg;
	hipFree(d_msg);
}
